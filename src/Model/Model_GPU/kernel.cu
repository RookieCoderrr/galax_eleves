
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU

#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)

__device__ float3 sub(float3 &a, float3 &b) {

  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);

}

__device__ float3 add(float3 &a, float3 &b) {

  return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);

}

__device__ float3 multi_1(float3 &a, float3 &b) {

  return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);

}

__device__ float3 multi_2(float3 &a, float b) {

  return make_float3(a.x * b, a.y * , a.z * b);

}


__device__ float3 multi_3(float3 &a, float b, float c) {

  return make_float3(a.x * b * c, a.y * b * c, a.z * b * c);

}

__global__ void compute_acc(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, float* massesGPU, int n_particles)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	for (int j = 0; j < n_particles; j++)
	{
		if(i != j)
		{
			const float3 diff = sub(positionsGPU[j] , positionsGPU[i]);
		
			float3  res = multi_1(diff, diff);
			float dij = res.x + res.y + res.z;
			
			if (dij < 1.0)
			{
				dij = 10.0;
			}
			else
			{
				dij = rsqrtf(dij);
				dij = 10.0 * (dij * dij * dij);
			}
			float3 acc = multi_3(diff, dij, massesGPU[j]);
			accelerationsGPU[i] = add(accelerationsGPU[i], acc);
			
		}
	}
}

__global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	velocitiesGPU[i] = add(velocitiesGPU[i], multi_2(accelerationsGPU[i], 2.0f));
	positionsGPU[i] = add(positionsGPU[i], multi_2(velocitiesGPU[i], 0.1f));

}

void update_position_cu(float3* positionsGPU, float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 128;
	int nblocks =  (n_particles + (nthreads -1)) / nthreads;

	compute_acc<<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU, n_particles);
	maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU);
}


#endif // GALAX_MODEL_GPU