
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU

#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)

__device__ float3 sub(const float3 &a, const float3 &b) {

  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);

}

__device__ float3 add(const float3 &a, const float3 &b) {

  return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);

}

__device__ float3 mul(const float3 &a, const float3 &b) {

  return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);

}

__device__ float3 mul(const float3 &a, float b) {

  return make_float3(a.x * b, a.y * b, a.z * b);

}


__device__ float3 mul(const float3 &a, float b, float c) {

  return make_float3(a.x * b * c, a.y * b * c, a.z * b * c);

}

__global__ void compute_acc(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, float* massesGPU, int n_particles)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	accelerationsGPU[i] = make_float3(0, 0, 0);
	if (i >= n_particles) {
		return;
	}
	for (int j = 0; j < n_particles; j++)
	{
		if(i != j)
		{
			const float3 diff = sub(positionsGPU[j] , positionsGPU[i]);
		
			float3  res = mul(diff, diff);
			float dij = res.x + res.y + res.z;
			
			if (dij < 1.0)
			{
				dij = 10.0;
			}
			else
			{
				dij = rsqrtf(dij);
				dij = 10.0 * (dij * dij * dij);
			}
			float3 acc = mul(diff, dij, massesGPU[j]);
			accelerationsGPU[i] = add(accelerationsGPU[i], acc);
		}
	}
}

__global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	velocitiesGPU[i] = add(velocitiesGPU[i], mul(accelerationsGPU[i], 2.0f));
	positionsGPU[i] = add(positionsGPU[i], mul(velocitiesGPU[i], 0.1f));
}

void update_position_cu(float3* positionsGPU, float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 128;
	int nblocks =  (n_particles + (nthreads -1)) / nthreads;

	compute_acc<<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU, n_particles);
	maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU);
}


#endif // GALAX_MODEL_GPU